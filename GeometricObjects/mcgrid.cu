#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include "ray.cuh"
#include "ray.h"
#include <hiprand/hiprand_kernel.h>
#include "CUDAhelpers.h"
#include "mcgrid.h"
//
//static __global__ bool inside_bb(CUDAreal3 &p0, CUDAreal3 &p1, CUDAreal3 &point) {
//	return
//		point.x > p0.x && point.x < p1.x &&
//		point.y > p0.y && point.y < p1.x &&
//		point.z > p0.z && point.z < p1.z;
//}
//
//static __global__ CUDAreal clamp(CUDAreal value, CUDAreal a, CUDAreal b) {
//	if (value < a) return a;
//	if (value > b) return b;
//	return value;
//}
//
//static __global__ void mcgrid_hit_kernel(
//	rayCU* rays, MCGridCUDA* gr, const int stride
//	)
//{
//	//Material* mat_ptr = sr.material_ptr;
//	int column = threadIdx.x + blockIdx.x * blockDim.x;
//	int row = threadIdx.y + blockIdx.y * blockDim.y;
//
//
//	rayCU ray = rays[column + stride * row];
//	MCGridCUDA grid = *gr;
//
//	int nx = grid.nx;
//	int ny = grid.ny;
//	int nz = grid.nz;
//
//
//	CUDAreal ox = ray.o.x;
//	CUDAreal oy = ray.o.y;
//	CUDAreal oz = ray.o.z;
//	CUDAreal dx = ray.d.x;
//	CUDAreal dy = ray.d.y;
//	CUDAreal dz = ray.d.z;
//	CUDAreal x0 = grid.p0.x;
//	CUDAreal y0 = grid.p0.y;
//	CUDAreal z0 = grid.p0.z;
//	CUDAreal x1 = grid.p1.x;
//	CUDAreal y1 = grid.p1.y;
//	CUDAreal z1 = grid.p1.z;
//	CUDAreal tx_min, ty_min, tz_min;
//	CUDAreal tx_max, ty_max, tz_max;
//	// the following code includes modifications from Shirley and Morley (2003)
//
//	CUDAreal a = 1.0 / dx;
//	if (a >= 0) {
//		tx_min = (x0 - ox) * a;
//		tx_max = (x1 - ox) * a;
//	}
//	else {
//		tx_min = (x1 - ox) * a;
//		tx_max = (x0 - ox) * a;
//	}
//
//	CUDAreal b = 1.0 / dy;
//	if (b >= 0) {
//		ty_min = (y0 - oy) * b;
//		ty_max = (y1 - oy) * b;
//	}
//	else {
//		ty_min = (y1 - oy) * b;
//		ty_max = (y0 - oy) * b;
//	}
//
//	CUDAreal c = 1.0 / dz;
//	if (c >= 0) {
//		tz_min = (z0 - oz) * c;
//		tz_max = (z1 - oz) * c;
//	}
//	else {
//		tz_min = (z1 - oz) * c;
//		tz_max = (z0 - oz) * c;
//	}
//
//	CUDAreal t0, t1;
//
//	if (tx_min > ty_min)
//		t0 = tx_min;
//	else
//		t0 = ty_min;
//
//	if (tz_min > t0)
//		t0 = tz_min;
//
//	if (tx_max < ty_max)
//		t1 = tx_max;
//	else
//		t1 = ty_max;
//
//	if (tz_max < t1)
//		t1 = tz_max;
//
//	if (t0 > t1)
//		return(false);
//
//
//	// initial cell coordinates
//
//	int ix, iy, iz;
//
//	if (inside_bb(grid.p0, grid.p1, ray.o)) {  			// does the ray start inside the grid?
//		ix = clamp((ox - x0) * nx / (x1 - x0), 0, nx - 1);
//		iy = clamp((oy - y0) * ny / (y1 - y0), 0, ny - 1);
//		iz = clamp((oz - z0) * nz / (z1 - z0), 0, nz - 1);
//	}
//	else {
//		CUDAreal3 p = ray.o + t0 * ray.d;  // initial hit point with grid's bounding box
//		ix = clamp((p.x - x0) * nx / (x1 - x0), 0, nx - 1);
//		iy = clamp((p.y - y0) * ny / (y1 - y0), 0, ny - 1);
//		iz = clamp((p.z - z0) * nz / (z1 - z0), 0, nz - 1);
//	}
//
//	// ray parameter increments per cell in the x, y, and z directions
//
//	CUDAreal dtx = (tx_max - tx_min) / nx;
//	CUDAreal dty = (ty_max - ty_min) / ny;
//	CUDAreal dtz = (tz_max - tz_min) / nz;
//
//	CUDAreal 	tx_next, ty_next, tz_next;
//	int 	ix_step, iy_step, iz_step;
//	int 	ix_stop, iy_stop, iz_stop;
//
//	if (dx > 0) {
//		tx_next = tx_min + (ix + 1) * dtx;
//		ix_step = +1;
//		ix_stop = nx;
//	}
//	else {
//		tx_next = tx_min + (nx - ix) * dtx;
//		ix_step = -1;
//		ix_stop = -1;
//	}
//
//	if (dx == 0.0) {
//		tx_next = kHugeValue;
//		ix_step = -1;
//		ix_stop = -1;
//	}
//
//
//	if (dy > 0) {
//		ty_next = ty_min + (iy + 1) * dty;
//		iy_step = +1;
//		iy_stop = ny;
//	}
//	else {
//		ty_next = ty_min + (ny - iy) * dty;
//		iy_step = -1;
//		iy_stop = -1;
//	}
//
//	if (dy == 0.0) {
//		ty_next = kHugeValue;
//		iy_step = -1;
//		iy_stop = -1;
//	}
//
//	if (dz > 0) {
//		tz_next = tz_min + (iz + 1) * dtz;
//		iz_step = +1;
//		iz_stop = nz;
//	}
//	else {
//		tz_next = tz_min + (nz - iz) * dtz;
//		iz_step = -1;
//		iz_stop = -1;
//	}
//
//	if (dz == 0.0) {
//		tz_next = kHugeValue;
//		iz_step = -1;
//		iz_stop = -1;
//	}
//
//	//    if (tx_next < 0) tx_next = kHugeValue;
//	//    if (ty_next < 0) ty_next = kHugeValue;
//	//    if (tz_next < 0) tz_next = kHugeValue;
//
//
//
//	// Test if there is a block face glued to the bounding box:
//
//	int block_id = grid.cells[ix + nx * iy + nx * ny * iz];
//	Point block_p0 = Point(x0 + nx * BLOCKLENGTH, y0 + ny * BLOCKLENGTH, z0 + nz * BLOCKLENGTH);
//	if (block_id != 0) {
//		real t_before = kHugeValue;
//
//		real tx_min_pp = tx_next - dtx;
//		real ty_min_pp = ty_next - dty;
//		real tz_min_pp = tz_next - dtz;
//
//		if (ix != 0 && ix != (nx - 1)) tx_min_pp = -kHugeValue;
//		if (iy != 0 && iy != (ny - 1)) ty_min_pp = -kHugeValue;
//		if (iz != 0 && iz != (nz - 1)) tz_min_pp = -kHugeValue;
//
//
//		if (tx_min_pp > ty_min_pp && tx_min_pp > tz_min_pp) {
//			sr.normal = Normal(-(real)ix_step, 0, 0);
//			sr.hdir = ix_step > 0 ? ShadeRec::South : ShadeRec::North;
//			t_before = tx_min_pp;
//		}
//		else if (ty_min_pp > tz_min_pp) {
//			sr.normal = Normal(0, -(real)iy_step, 0);
//			sr.hdir = iy_step > 0 ? ShadeRec::Bottom : ShadeRec::Top;
//			t_before = ty_min_pp;
//
//		}
//		else {
//			sr.normal = Normal(0, 0, -(real)iz_step);
//			sr.hdir = iz_step > 0 ? ShadeRec::West : ShadeRec::East;
//			t_before = tz_min_pp;
//
//		}
//		if (block_ptr->block_hit(ray, block_p0, t_before, sr)) {
//			t = t_before;
//
//
//			return (true);
//		}
//	}
//
//
//
//	// traverse the grid
//	t = kHugeValue;
//	real t_before = kHugeValue;
//
//	while (true) {
//		//        MCBlock* block_ptr = cells[ix + nx * iy + nx * ny * iz];
//
//		if (tx_next < ty_next && tx_next < tz_next) {
//			//real tmin = tx_next - kEpsilon;
//			//Material* mptr = sr.material_ptr;
//			sr.normal = Normal(-(real)ix_step, 0, 0);
//			sr.hdir = ix_step > 0 ? ShadeRec::South : ShadeRec::North;
//			sr.t_Before = t_before;
//			t_before = tx_next;
//			tx_next += dtx;
//			ix += ix_step;
//			if (ix == ix_stop) {
//				sr.material_ptr = mat_ptr;
//				return (false);
//			}
//
//
//			MCBlock* block_ptr = _w->blocklist[cells[ix + nx * iy + nx * ny * iz]];
//			Point block_p0 = Point(x0 + nx * BLOCKLENGTH, y0 + ny * BLOCKLENGTH, z0 + nz * BLOCKLENGTH);
//
//			if (block_ptr && block_ptr->block_hit(ray, block_p0, t_before, sr)) {
//				t = t_before;
//
//
//				return (true);
//			}
//			//sr.material_ptr = mptr;
//
//		}
//		else {
//			if (ty_next < tz_next) {
//				//Material* mptr = sr.material_ptr;
//				sr.normal = Normal(0.0, -(real)iy_step, 0);
//				sr.hdir = iy_step > 0 ? ShadeRec::Bottom : ShadeRec::Top;
//				sr.t_Before = t_before;
//				t_before = ty_next;
//				ty_next += dty;
//				iy += iy_step;
//				if (iy == iy_stop) {
//					sr.material_ptr = mat_ptr;
//					return (false);
//				}
//
//				MCBlock* block_ptr = _w->blocklist[cells[ix + nx * iy + nx * ny * iz]];
//				Point block_p0 = Point(x0 + nx * BLOCKLENGTH, y0 + ny * BLOCKLENGTH, z0 + nz * BLOCKLENGTH);
//
//
//				if (block_ptr && block_ptr->block_hit(ray, block_p0, t_before, sr)) {
//					//material_ptr = object_ptr->get_material();
//					t = t_before;
//					//t = ty_next;
//					return (true);
//				}
//				//sr.material_ptr = mptr;
//				//mat_ptr
//
//			}
//			else {
//				//Material* mptr = sr.material_ptr;
//				sr.normal = Normal(0.0, 0.0, -(real)iz_step);
//				sr.hdir = iz_step > 0 ? ShadeRec::West : ShadeRec::East;
//				sr.t_Before = t_before;
//				t_before = tz_next;
//				tz_next += dtz;
//				iz += iz_step;
//				if (iz == iz_stop) {
//					sr.material_ptr = mat_ptr;
//					return (false);
//				}
//
//				MCBlock* block_ptr = _w->blocklist[cells[ix + nx * iy + nx * ny * iz]];
//				Point block_p0 = Point(x0 + nx * BLOCKLENGTH, y0 + ny * BLOCKLENGTH, z0 + nz * BLOCKLENGTH);
//
//
//				//material_ptr = sr.material_ptr;
//				if (block_ptr && block_ptr->block_hit(ray, block_p0, t_before, sr)) {
//					//material_ptr = object_ptr->get_material();
//					//sr.material_ptr = material_ptr;
//					t = t_before;
//					// t = tz_next;
//					return (true);
//				}
//				//sr.material_ptr = mptr;
//
//			}
//		}
//	}
//}